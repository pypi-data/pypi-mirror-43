#include <ATen/ATen.h>

#include <hipsparse.h>

#define CSRGEMM(TYPE, ...)                                                     \
  [&] {                                                                        \
    const at::Type &the_type = TYPE;                                           \
    switch (the_type.scalarType()) {                                           \
    case at::ScalarType::Float: {                                              \
      using scalar_t = float;                                                  \
      return hipsparseScsrgemm(__VA_ARGS__);                                    \
    }                                                                          \
    case at::ScalarType::Double: {                                             \
      using scalar_t = double;                                                 \
      return hipsparseDcsrgemm(__VA_ARGS__);                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '%s'", the_type.toString());               \
    }                                                                          \
  }()

static hipsparseHandle_t cusparse_handle = 0;

static void init_cusparse() {
  if (cusparse_handle == 0) {
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
  }
}

std::tuple<at::Tensor, at::Tensor>
spspmm_cuda(at::Tensor indexA, at::Tensor valueA, at::Tensor indexB,
            at::Tensor valueB, int m, int k, int n) {
  hipSetDevice(indexA.get_device());
  init_cusparse();

  indexA = indexA.contiguous();
  valueA = valueA.contiguous();
  indexB = indexB.contiguous();
  valueB = valueB.contiguous();

  auto nnzA = valueA.size(0);
  auto nnzB = valueB.size(0);

  indexA = indexA.toType(at::kInt);
  indexB = indexB.toType(at::kInt);

  // Convert A to CSR format.
  auto row_ptrA = at::empty(m + 1, indexA.type());
  hipsparseXcoo2csr(cusparse_handle, indexA[0].data<int>(), nnzA, k,
                   row_ptrA.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);
  auto colA = indexA[1];
  hipMemcpy(row_ptrA.data<int>() + m, &nnzA, sizeof(int),
             hipMemcpyHostToDevice);

  // Convert B to CSR format.
  auto row_ptrB = at::empty(k + 1, indexB.type());
  hipsparseXcoo2csr(cusparse_handle, indexB[0].data<int>(), nnzB, k,
                   row_ptrB.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);
  auto colB = indexB[1];
  hipMemcpy(row_ptrB.data<int>() + k, &nnzB, sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  int nnzC;
  auto row_ptrC = at::empty(m + 1, indexB.type());
  hipsparseXcsrgemmNnz(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnzA,
                      row_ptrA.data<int>(), colA.data<int>(), descr, nnzB,
                      row_ptrB.data<int>(), colB.data<int>(), descr,
                      row_ptrC.data<int>(), &nnzC);
  auto colC = at::empty(nnzC, indexA.type());
  auto valueC = at::empty(nnzC, valueA.type());

  CSRGEMM(valueC.type(), cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
          HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnzA,
          valueA.data<scalar_t>(), row_ptrA.data<int>(), colA.data<int>(),
          descr, nnzB, valueB.data<scalar_t>(), row_ptrB.data<int>(),
          colB.data<int>(), descr, valueC.data<scalar_t>(),
          row_ptrC.data<int>(), colC.data<int>());

  auto rowC = at::empty(nnzC, indexA.type());
  hipsparseXcsr2coo(cusparse_handle, row_ptrC.data<int>(), nnzC, m,
                   rowC.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);

  auto indexC = at::stack({rowC, colC}, 0).toType(at::kLong);

  return std::make_tuple(indexC, valueC);
}
