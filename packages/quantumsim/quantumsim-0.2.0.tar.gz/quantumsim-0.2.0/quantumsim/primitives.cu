/*This file is part of quantumsim. (https://gitlab.com/quantumsim/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE or https://www.gnu.org/licenses/gpl.txt*/

#include <hip/hip_runtime.h>

//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
//run with a 2d grid of total size (2**no_qubits)^2
__global__ void bit_to_pauli_basis(double *complex_dm, unsigned int mask, unsigned int no_qubits) {
    const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

    const double sqrt2 =  0.70710678118654752440;
    //const double sqrt2 =  1;

    if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;

    int b_addr = ((x|mask)<<no_qubits | (y&~mask)) << 1;
    int c_addr = ((x&~mask)<<no_qubits | (y|mask)) << 1;

    if (x&mask && (~y&mask)){
        double b = complex_dm[b_addr];
        double c = complex_dm[c_addr];
        complex_dm[b_addr] = (b+c)*sqrt2;
        complex_dm[c_addr] = (b-c)*sqrt2;
    }
    if ((~x&mask) && (y&mask)){
        b_addr+=1;
        c_addr+=1;
        double b = complex_dm[b_addr];
        double c = complex_dm[c_addr];
        complex_dm[b_addr] = (b+c)*sqrt2;
        complex_dm[c_addr] = (b-c)*sqrt2;
    }
}


//pauli_reshuffle
//this function collects the values from a complex density matrix in (0, x, iy, 1) basis
//and collects the real or values only; furthermore it rearranges the address bit order
//from (d_state_bits, d_state_bits) to
// (alpha_d, alpha_d-1, ..., alpha_0) where alpha = (00, 01, 10, 11) for 0, x, y, 1
//if direction = 0, the copy is performed from complex to real, otherwise from real to complex
__global__ void pauli_reshuffle(double *complex_dm, double *real_dm, unsigned int no_qubits, unsigned int direction) {

    const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

    if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;


    //do we need imaginary part? That is the case if we have an odd number of bits for y in our adress (bit in y is 1, bit in x is 0)
    unsigned int v = ~x & y;


    unsigned int py = 0;
    while (v) {
        py += v&1;
        v >>= 1;
    }

    py = py & 0x3;

    //short version: while (v>1) { v = (v >> 1) ^ v ;}
    //bit bang version
    /*v ^= v >> 1;*/
    /*v ^= v >> 2;*/
    /*v = (v & 0x11111111U) * 0x11111111U;*/
    /*v = (v >> 28) & 1;*/

    const unsigned int addr_complex = (((x << no_qubits) | y) << 1) + (py&1);


    //the adress in pauli basis is obtained by interleaving
    unsigned int addr_real = 0;
    for (int i = 0; i < 16; i++) {
          addr_real |= (x & 1U << i) << i | (y & 1U << i) << (i + 1);
    }


    if(direction == 0) {
        real_dm[addr_real] = ((py==3 || py==2)? -1 : 1)*complex_dm[addr_complex];
    }
    else {
        complex_dm[addr_complex] = ((py==3 || py == 2)? -1 : 1)*real_dm[addr_real];
    }
}



__global__ void two_qubit_general_ptm(double *dm, double *ptm_g,
        unsigned int dim_a, unsigned int stride_a,
        unsigned int dim_b, unsigned int stride_b,
        unsigned int dim_rho) {

    const unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;


    // external memory required: (blockDim.x + dim_a*dim_b) double floats
    extern __shared__ double ptm[];
    double *data = &ptm[dim_a*dim_b];

    // load ptm to shared memory (ptm should be smaller than block, but in case it is not, loop here)
    for(int i=0; i < dim_a*dim_b; i+=blockDim.x) {
        if(i+threadIdx.x < dim_a*dim_b) {
            ptm[i+threadIdx.x] = ptm_g[i+threadIdx.x];
        }
    }

    if (idx >= dim_rho) return;

    //adress calculation
    //the index is of the form idx = X Y Z ib ia,
    //where the address is of the form addr = X ib Y ia Z
    //this integer arithmetic is possibly is quite slow. one might want to do it in float instead

    unsigned int i = idx;
    unsigned int reduced_stride_of_y = stride_b/(stride_a*dim_a);
    unsigned int idx_a = i % dim_a;
    i = i / dim_a;
    unsigned int idx_b = i % dim_b;
    i = i / dim_b;
    unsigned int     z = i % (stride_a);
    i = i / stride_a;
    unsigned int     y = i % reduced_stride_of_y;
    unsigned int     x = i / reduced_stride_of_y;

    unsigned int addr = z + stride_a*idx_a + stride_a*dim_a*y + stride_b*idx_b + stride_b*dim_b*x;

    //fetch data to memory
    //data[threadIdx.x] = dm[addr];
    __syncthreads();


    /*int row = idx_b*dim_b + idx_a;//000 ib ia;*/
    /*int offset = idx - row;          //x y z00;*/

    double acc=1;
    /*for(int i=0; i<dim_a*dim_b; i++) {*/
        /*acc += ptm[dim_a*dim_b*row + i]*data[offset+i];*/
    /*}*/

    //upload back to global memory
    __syncthreads();
    dm[addr] = acc;
}


// apply a 4x4 pauli transfer matrix (in 0, x, y, 1 basis)
// to the specified qubit
__global__ void single_qubit_ptm(double *dm, double *ptm_g,  unsigned int bit, unsigned int no_qubits) {
    const unsigned int x = threadIdx.x;
    const unsigned int high_x = blockIdx.x * blockDim.x;

    if (high_x + x >= (1 << (2*no_qubits))) return;

    //the two lowest bits of thread id are used to index the target bit,
                                                //      xx <- target bit
    int high_mask = ~ ( (1 << (2*bit+2)) - 1 ); // 1111100000000
    int low_mask  = ~high_mask & (~0x3);        // 0000011111100

    int pos = high_x | x;
    int global_from = (pos & high_mask) | ((pos & 0x3) << (2*bit)) | ((pos & low_mask)>>2);

    extern __shared__ double ptm[];
    double *data = &ptm[16]; //need blockDim.x double floats

    //first fetch the transfer matrix to shared memory
    if(x < 16) ptm[x] = ptm_g[x];

    if(no_qubits < 2) { //what a boring situation
        ptm[x+4] = ptm_g[x+4];
        ptm[x+8] = ptm_g[x+8];
        ptm[x+12] = ptm_g[x+12];
    }

    //fetch block to shared memory
    data[x] = dm[global_from];
    __syncthreads();

    //do calculation

    int row = x & 0x3;
    int idx = x & ~0x3;

    double acc = 0;

    acc += ptm[4*row    ] * data[idx    ];
    acc += ptm[4*row + 1] * data[idx + 1];
    acc += ptm[4*row + 2] * data[idx + 2];
    acc += ptm[4*row + 3] * data[idx + 3];

    //upload back to global memory
    __syncthreads();
    dm[global_from] = acc;
}


__global__ void two_qubit_ptm(double *dm, double *ptm_g, unsigned int bit0, unsigned int bit1, unsigned int no_qubits) {
    const unsigned int x = threadIdx.x;
    const unsigned int high_x = blockIdx.x * blockDim.x;



    extern __shared__ double ptm[];
    double *data = &ptm[256]; //need blockDim.x double floats

    // the lowest to bits of x are used to address bit0, the next two are used to address bit1
    // global address = <- pos =
    // aaaxxbbbbyycccc  <- aaabbbbccccxxyy

    int higher_bit = max(bit0, bit1);
    int lower_bit = min(bit0, bit1);
    int high_mask = ~ ( (1 << (2*higher_bit+2)) - 1 ); //a mask (of pos)
    int mid_mask = (~ ( (1 << (2*lower_bit + 4)) - 1)) & (~high_mask);  //b mask
    int low_mask  = ~(high_mask | mid_mask) & (~0xf);  //c mask

    int pos = high_x | x;
    int global_from =
              (pos & high_mask)
            | ((pos & mid_mask) >> 2)
            | ((pos & low_mask) >> 4)
            | ((pos & 0x3) << (2 * bit0))
            | (((pos & 0xc) >>2)  << (2 * bit1));

    //fetch ptm to shared memmory
    //need to fetch several values per thread if blockDim.x is less than 256 (only for small dms...)
    for(int i=0; i < 256; i+=blockDim.x) {
        if(i+x < 256) {
            ptm[i+x] = ptm_g[i+x];
        }
    }
    if (high_x + x >= (1 << (2*no_qubits))) return;


    //fetch data block to shared memory
    data[x] = dm[global_from];
    __syncthreads();

    unsigned int row = x & 0xf;
    unsigned int idx = x & ~0xf;

    double acc=0;
    for(int i=0; i<16; i++) {
        acc += ptm[16*row + i]*data[idx+i];
    }


    __syncthreads();
    dm[global_from] = acc;

}


//copy the two diagonal blocks of one ancilla into reduced density matrices
//the qubit index is passed as an integer, not as a bitmask!
__global__ void dm_reduce(double *dm, unsigned int bit, double *dm0, unsigned int state,
        unsigned int no_qubits) {

    const int addr = blockIdx.x*blockDim.x + threadIdx.x;

    if(addr >= (1<< (2*no_qubits))) return;

    const int low_mask = (1 << (2*bit))-1;      //0000011111
    const int high_mask = (~low_mask) << 2;     //1110000000

    if(((addr >> (2*bit)) & 0x3) == state*0x3) {
        dm0[ (addr & low_mask) | ((addr & high_mask) >> 2) ] = dm[addr];
    }
}



//get_diagonal kernel
//copy the diagonal elements to out, in order to do effective
//calculation of subtraces.
//run over a 1x9 grid!
__global__ void get_diag(double *dm9, double *out, unsigned int no_qubits) {
    int x = (blockIdx.x *blockDim.x) + threadIdx.x;

    if (x >= (1 << no_qubits)) return;
    unsigned int addr_real = 0;
    for (int i = 0; i < 16; i++) {
          addr_real |= (x & 1U << i) << i | (x & 1U << i) << (i + 1);
    }
    out[x] = dm9[addr_real];
}

//trace kernel. Calculate the sum of a diagonal, must run in one block!
//shared memory: 2**no_qubits doubles
//if bit is positive or zero, diag[0] and diag[1] will hold the partial traces of this bit being one/zero (!note the switch)
//if bit is -1, diag[0] will hold the full trace.
__global__ void trace(double *diag, int bit) {
    unsigned int x = threadIdx.x;
    unsigned int mask = 0;

    if(bit >= 0) {
        mask = 1 << bit;
    }

    extern __shared__ double s_diag[];
    s_diag[x] = diag[x];
    __syncthreads();

    double a;

    for(unsigned int i=1; i < blockDim.x; i <<= 1) {
        if(i != mask && i <= x) {
            a = s_diag[x-i];

        }
        __syncthreads();
        if(i != mask && i <= x) {
            s_diag[x] += a;
        }
        __syncthreads();
    }

    __syncthreads();
    //copy result back
    if(x == 0) {
        diag[blockIdx.x] = s_diag[blockDim.x - 1];
        return;
    }
    if(x == 1 && bit >= 0) {
        diag[blockIdx.x + 1] = s_diag[blockDim.x - 1 - mask];
        return;
    }
}

//swap kernel
//exchange two qubits. The only purpose of this kernel is to arrange a certain qubit as to be the most significant so that
//projection is trivial. Actual swap gates should be implemented by relabeling!
__global__ void swap(double *dm, unsigned int bit1, unsigned int bit2, unsigned int no_qubits) {
    unsigned int addr = threadIdx.x + blockDim.x*blockIdx.x;

    if (addr >= (1<<2*no_qubits)) return;

    unsigned int bit1_mask = (0x3 << (2*bit1));
    unsigned int bit2_mask = (0x3 << (2*bit2));

    unsigned int addr2 = ( addr & ~(bit1_mask | bit2_mask)) |
        ((addr & bit1_mask) << (2*(bit2 - bit1))) |
        ((addr & bit2_mask) >> (2*(bit2 - bit1)));

    double t;
    if (addr > addr2) {
        t = dm[addr2];
        dm[addr2] = dm[addr];
        dm[addr] = t;
    }
}
